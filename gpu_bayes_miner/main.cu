#include <iostream>
#include <hip/hip_runtime.h>

#define N 2  // Change as needed, e.g., 16, 32

// CUDA Kernel
__global__ void MatAdd(float A[N][N], float B[N][N], float C[N][N]) {
    int i = threadIdx.x;
    int j = threadIdx.y;
    if (i < N && j < N)
        C[i][j] = A[i][j] + B[i][j];
}

int main() {
    float A[N][N], B[N][N], C[N][N];
    float (*d_A)[N], (*d_B)[N], (*d_C)[N];

    // Initialize input arrays
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++) {
            A[i][j] = i;
            B[i][j] = j;
        }

    // Allocate memory on the device
    hipMalloc((void**)&d_A, N * N * sizeof(float));
    hipMalloc((void**)&d_B, N * N * sizeof(float));
    hipMalloc((void**)&d_C, N * N * sizeof(float));

    // Copy data to device
    hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Kernel call
    dim3 threadsPerBlock(N, N);
    MatAdd<<<1, threadsPerBlock>>>(d_A, d_B, d_C);

    // Copy result back to host
    hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Print result
    std::cout << "Result matrix C:\n";
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++)
            std::cout << C[i][j] << " ";
        std::cout << "\n";
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
